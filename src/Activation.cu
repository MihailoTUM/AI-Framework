#include "hip/hip_runtime.h"
#include <iostream>
#include "Tensor.cu"

__global__ void reluGPU(float *A, float *C, int rows, int cols) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int size = rows * cols;

    if(idx < size) {
        if(A[idx] < 0) {
            C[idx] = 0.0f;
        } else {
            C[idx] = A[idx];
        }
    };
};


class Activation {
    private:
        char function;
    
    public:
    Activation(char func = 'R') {
        function = func;
    }

    char getFunction() { return function; };

    void reluCPU (float *A, int rows, int cols) {
        for(int i = 0; i < rows* cols; i++) {
            if(A[i] < 0) {
                A[i] = 0;
            };
        };
    };

    Tensor forward(const Tensor& other) {
        if(function == 'R') {
            return relu(other);
        }
        return other;
    };

    Tensor relu(const Tensor& input) {
        int rows = input.getRows();
        int cols = input.getCols();
        Tensor result(rows, cols, input.getDevice(), false);

        if(input.getDevice() == 'C') {
            std::cout << "HAPPENS ON CPU" << std::endl;
            reluCPU(input.getMatrix(), rows, cols);
            return result;
        }   
        else {
            std::cout << "HAPPENS ON GPU" << std::endl;
            size_t size = rows * cols * sizeof(float);
            
            float *d_A, *d_C;
            hipMalloc(&d_A, size);
            hipMalloc(&d_C, size);

            hipMemcpy(d_A, input.getMatrix(), size, hipMemcpyHostToDevice);

            int threads = 256;
            int blocks = (rows * cols + threads - 1)/threads;

            reluGPU<<<blocks, threads>>>(d_A, d_C, rows, cols);

            hipMemcpy(result.getMatrix(), d_C, size, hipMemcpyDeviceToHost);

            hipFree(d_A);
            hipFree(d_C);
            
            return result;
        }   
    };

    Tensor sigmoid();

    Tensor tanh();
};

int main() {

    Tensor cuda(3, 3, 'G', true);
    cuda.print();

    std::cout << "\n";

    Activation ac = ('R');
    Tensor relu = ac.forward(cuda);
    relu.print();

    return 0;
}