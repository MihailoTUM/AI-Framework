#include "hip/hip_runtime.h"
#include "Tensor.h"
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>

__global__ void addMatrixGPU(float* A, float* B, float* C, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = rows * cols;

    if(idx < size) {
        C[idx] = A[idx] + B[idx];
    };
}

__global__ void matmulGPU(float *A, float *B, float * C, int nA, int nB, int nC) {
    int size = nA * nC;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = idx / nC;
    int j = idx % nC;


    if (idx < size) {
        float sum = 0.0f;
        for(int run = 0; run < nB; run++) {
            sum += A[run + i * nB] * B[run * nC + j];
        }
        C[i * nC + j] = sum;
    };
};

__global__ void scalarGPU(float *A, float scalar, float* C, int rows, int cols) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int size = rows * cols;
    if(idx < size) {
        C[idx] = scalar * A[idx];
    }
}

__global__ void addBroadcastGPU(float *A, float *B, float *C, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = rows * cols;
    int i = idx / cols;
    int j = idx % cols;

    if(idx < size) {
        C[i * cols + j] = A[i * cols + j] + B[j];
    }
}

__global__ void sumGPU(float *A, int axis, float *C, int rows, int cols) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int size = rows * cols;

    int i = idx / cols; 
    int j = idx % cols;

    if(idx >= size) return;

        if(axis == 0){
            // (5, 3) -> (1, 3);
            atomicAdd(&C[j], A[i * cols + j]);
        }
        else {
            //axis == 1;
            atomicAdd(&C[i], A[i * cols + j]);
        }
    
}; 

__global__ void meanGPU(float* A, int axis, float *C, int rows, int cols) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int size = rows * cols;

    int i = idx / cols;
    int j = idx % cols;

    if(idx >= size) return;

    if(axis == 0) {
        atomicAdd(&C[i], A[i * cols + j]/rows);
    }
    else {
        atomicAdd(&C[i], A[i * cols + j]/cols);
    };
};

Tensor::Tensor(int nRows, int nCols, char nDevice, bool random) {
    rows = nRows;
    cols = nCols;
    device = nDevice;
    matrix = new float[rows * cols];

    if(random) {
        initMatrixRandom();
    }
    else {
        initMatrixToZeros();
    }
};

Tensor::~Tensor() {
    delete[] matrix;
}

int Tensor::getRows() const {
    return rows;
}

int Tensor::getCols() const {
    return cols;
}

float Tensor::getValue(int row, int col) const {
    return matrix[row * cols + col];
};

float* Tensor::getMatrix() const {
    return matrix;
}

char Tensor::getDevice() const {
    return device;
}

void Tensor::setValue(int row, int col, float value) {
    matrix[row * cols + col] = value;
};

void Tensor::initMatrixToZeros() {
        for(int i = 0; i < rows; i++) {
            for(int j = 0; j < cols; j++) {
                setValue(i, j, 0.0f);
            };
        };
    };

void Tensor::initMatrixRandom() {

        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
        
        for(int i = 0; i < rows; i++) {
            for(int j = 0; j < cols; j++) {
                setValue(i, j, dist(gen));
            };
        };
    };

void Tensor::print() const {
        for(int i = 0; i < rows; i++) {
            for(int j = 0; j < cols; j++) {
                std::cout << getValue(i, j) << " ";
            };
            std::cout << "\n";
        };
    }

void Tensor::addMatrixCPU(float *A, float *B, float *C, int rows, int cols) const {
        for(int i = 0; i < rows; i++) {
            for(int j = 0; j < cols; j++) {
                C[i * cols + j] = A[i * cols + j] + B[i * cols + j];
            }
        }
    }

void Tensor::matmulCPU(float *A, float* B, float *C, int rows, int mix, int cols) const {
        for(int x = 0; x < rows; x++) {
            for(int y = 0; y < cols; y++) {
                float value = 0;
                    for(int k = 0; k < cols; k++) {
                        value += A[cols * x + k] * B[cols * k + y];
                    }
                C[x * cols + y] = value;
            }     
        }
    }

void Tensor::scalarCPU(float* A, float scalar, float *C, int rows, int cols) const{
        for(int i = 0; rows; i++) {
            for(int j = 0; j < cols; j++) {
                A[i * cols + j] = scalar * C[i * cols + j];
            }
        }
    }

void Tensor::addBroadcastCPU(float *A, float *B, float* C, int rows, int cols) const {
        for(int i = 0; i < rows; i++) {
            for(int j = 0; j < cols; j++) {
                C[i * cols + j] = A[i * cols + j] + B[j];
            }
        } 
    };

Tensor Tensor::operator+(const Tensor& other) const {
        // allow broadcasting
        if(rows == other.rows && cols == other.cols) {
            if(device == other.device) {
                Tensor result(rows, cols, device, false);
                if(device == 'C') {
                    std::cout << "HAPPENS ON CPU \n";
                    addMatrixCPU(getMatrix(), other.getMatrix(), result.getMatrix(), rows, cols);
                }
                else if(device == 'G') {
                    std::cout << "HAPPENS ON GPU \n";
                    size_t size = rows * cols * sizeof(float);

                    float *d_A, *d_B, *d_C;
                    hipMalloc(&d_A, size);
                    hipMalloc(&d_B, size);
                    hipMalloc(&d_C, size);

                    hipMemcpy(d_A, getMatrix(), size, hipMemcpyHostToDevice);
                    hipMemcpy(d_B, other.getMatrix(), size, hipMemcpyHostToDevice);
                
                    int threads = 256;
                    int blocks = (rows * cols + threads - 1)/threads;
                    addMatrixGPU<<<blocks, threads>>>(d_A, d_B, d_C, other.getRows(), other.getCols());

                    hipMemcpy(result.getMatrix(), d_C, size, hipMemcpyDeviceToHost);

                    hipFree(d_A);
                    hipFree(d_B);
                    hipFree(d_C);
                }
                else {
                    throw std::invalid_argument("Invalid");
                };
                return result;
            }
            else {
                throw std::invalid_argument("Not on the same device!");
            }
        }
        else if(cols == other.cols && other.rows == 1) {
            if(getDevice() == other.getDevice()) {
                Tensor result(rows, cols, getDevice(), false);
                if(device == 'C') {
                    std::cout << "HAPPENS ON CPU";
                    addBroadcastCPU(getMatrix(), other.getMatrix(), result.getMatrix(), rows, cols);
                    return result;
                }
                else if(device == 'G') {
                    std::cout << "HAPPENS ON GPU \n";
                    size_t sizeA = rows * cols * sizeof(float);
                    size_t sizeB = other.cols * sizeof(float);

                    float *d_A, *d_B, *d_C;
                    hipMalloc(&d_A, sizeA);
                    hipMalloc(&d_B, sizeB);
                    hipMalloc(&d_C, sizeA);

                    hipMemcpy(d_A, getMatrix(), sizeA, hipMemcpyHostToDevice);
                    hipMemcpy(d_B, other.getMatrix(), sizeB, hipMemcpyHostToDevice);
                
                    int threads = 256;
                    int blocks = (rows * cols + threads - 1)/threads;
                    addBroadcastGPU<<<blocks, threads>>>(d_A, d_B, d_C, getRows(), getCols());

                    hipMemcpy(result.getMatrix(), d_C, sizeA, hipMemcpyDeviceToHost);

                    hipFree(d_A);
                    hipFree(d_B);
                    hipFree(d_C);

                    return result;
                }   
                else {
                    throw std::invalid_argument("Invalid arguments passed!");
                }
            }
            else {
                throw std::invalid_argument("Invalid argument!");
            }
        }
        else {
            throw std::invalid_argument("Invalid dimensions!");
        }
    };

Tensor Tensor::operator*(const Tensor& other) const {
        if(cols == other.rows) {
            if(device == other.device) {
                Tensor result(rows, other.cols, device, false);
                if(device == 'C') {
                    std::cout << "HAPPENS ON CPU \n";
                    matmulCPU(getMatrix(), other.getMatrix(), result.getMatrix(), rows, cols, other.cols);
                }
                else if(device == 'G') {
                    std::cout << "HAPPENS ON GPU \n";
                    size_t size_A = rows * cols * sizeof(float);
                    size_t size_B = other.getRows() * other.getCols() * sizeof(float);
                    size_t size_C = rows * getCols() * sizeof(float);

                    float *d_A, *d_B, *d_C;
                    hipMalloc(&d_A, size_A);
                    hipMalloc(&d_B, size_B);
                    hipMalloc(&d_C, size_C);

                    hipMemcpy(d_A, getMatrix(), size_A, hipMemcpyHostToDevice);
                    hipMemcpy(d_B, other.getMatrix(), size_B, hipMemcpyHostToDevice);
                
                    int threads = 256;
                    int blocks = (rows * cols + threads - 1)/threads;
                    matmulGPU<<<blocks, threads>>>(d_A, d_B, d_C, getRows(), getCols(), other.getCols());

                    hipMemcpy(result.getMatrix(), d_C, size_C, hipMemcpyDeviceToHost);

                    hipFree(d_A);
                    hipFree(d_B);
                    hipFree(d_C);
                }
                else {
                    throw std::invalid_argument("Invalid");
                };
                return result;
            }
            else {
                throw std::invalid_argument("Not on the same device!");
            }
        }
        else {
            throw std::invalid_argument("Invalid dimensions!");
        }
    }

Tensor Tensor::operator* (float scalar) const {
        Tensor result(getRows(), getCols(), getDevice(), false);
        if(getDevice() == 'C') {
            std::cout << "HAPPENS ON CPU";
            scalarCPU(getMatrix(), scalar, result.getMatrix(), getRows(), getCols());
        }
        else if(getDevice() == 'G'){
                std::cout << "HAPPENS ON GPU \n";
                size_t size = rows * cols * sizeof(float);

                float *d_A, *d_C;
                hipMalloc(&d_A, size);
                hipMalloc(&d_C, size);

                hipMemcpy(d_A, getMatrix(), size, hipMemcpyHostToDevice);
                
                int threads = 256;
                int blocks = (rows * cols + threads - 1)/threads;
                scalarGPU<<<blocks, threads>>>(d_A, scalar, d_C, getRows(), getCols());

                hipMemcpy(result.getMatrix(), d_C, size, hipMemcpyDeviceToHost);

                hipFree(d_A);
                hipFree(d_C);
        }
        else {
            throw std::invalid_argument("Invalid arguments!");
        };
        return result;
    };

Tensor Tensor::operator-() const {
        Tensor result(rows, cols, device, false);
        for(int i = 0; i < rows; i++) {
            for(int j = 0; j < cols; j++) {
                result.getMatrix()[i * cols + j] = getValue(i, j);
            }
        }
        return result;
    }
Tensor Tensor::sum(int axis) const {
        if(axis == 0) {
            if(device == 'C') {
            Tensor result(1, getCols(), getDevice(), false);
             for(int k = 0; k < this->cols; k++) {
                float sum = 0;
                for(int i = 0; i < this->rows; i++) {
                    sum += this->matrix[i * this->cols + k];
                }
                result.setValue(0, k, sum);
            }
            return result;
            } else if(device == 'G') {
                Tensor result(1, getCols(), getDevice(), false);
                std::cout << "HAPPENS ON GPU \n";
                size_t sizeA = getRows() * getCols() * sizeof(float);
                size_t sizeC = getCols() * sizeof(float);

                float *d_A, *d_C;
                hipMalloc(&d_A, sizeA);
                hipMalloc(&d_C, sizeC);

                hipMemcpy(d_A, getMatrix(), sizeA, hipMemcpyHostToDevice);
                
                int threads = 256;
                int blocks = (rows * cols + threads - 1)/threads;
                sumGPU<<<blocks, threads>>>(d_A, axis, d_C, getRows(), getCols());

                hipMemcpy(result.getMatrix(), d_C, sizeC, hipMemcpyDeviceToHost);

                hipFree(d_A);
                hipFree(d_C);

                return result;
            }
            else {
                throw std::invalid_argument("Invalid device argument!");
            }
        }   
        else if (axis == 1) {
            if(device == 'C') {
                Tensor result(getRows(), 1, getDevice(), false);
                for(int k = 0; k < this->rows; k++) {
                    float sum = 0;
                    for(int i = 0; i < this->cols; i++) {
                        sum += this->matrix[k * this->cols + i];
                    }
                    result.setValue(k, 0, sum);
                }
                return result;
            } 
            else if(device == 'G') {
                Tensor result(getRows(), 1, getDevice(), false);
                std::cout << "HAPPENS ON GPU \n";
                size_t sizeA = getRows() * getCols() * sizeof(float);
                size_t sizeC = getRows() * sizeof(float);

                float *d_A, *d_C;
                hipMalloc(&d_A, sizeA);
                hipMalloc(&d_C, sizeC);

                hipMemcpy(d_A, getMatrix(), sizeA, hipMemcpyHostToDevice);
                
                int threads = 256;
                int blocks = (rows * cols + threads - 1)/threads;
                sumGPU<<<blocks, threads>>>(d_A, axis, d_C, getRows(), getCols());

                hipMemcpy(result.getMatrix(), d_C, sizeC, hipMemcpyDeviceToHost);

                hipFree(d_A);
                hipFree(d_C);

                return result;
            }
            else {
                throw std::invalid_argument("Invalid device argument!");
            }
        }
        else {
            throw std::invalid_argument("Invalid axis > 1");
        }
    };
Tensor Tensor::mean(int axis) const {
        if(axis == 0) {
            Tensor result(1, getCols(), getDevice(), false);
             for(int k = 0; k < this->cols; k++) {
                float sum = 0;
                for(int i = 0; i < this->rows; i++) {
                    sum += this->matrix[i * this->cols + k];
                }
                result.setValue(0, k, sum/rows);
            }
            return result;
        }   
        else if (axis == 1) {
            Tensor result(getRows(), 1, getDevice(), false);
            for(int k = 0; k < this->rows; k++) {
                float sum = 0;
                for(int i = 0; i < this->cols; i++) {
                    sum += this->matrix[k * this->cols + i];
                }
                result.setValue(k, 0, sum/cols);
            }
            return result;
        }
        else {
            throw std::invalid_argument("Invalid axis > 1");
        }
    };
Tensor Tensor::transpose() {
    Tensor result(getCols(), getRows(), device, false);
        for(int i = 0; i < rows; i++) {
            for(int j = 0; j < cols; j++) {
                result.getMatrix()[j * rows + i] = matrix[i * cols + j];
            }
        }
        return result;
};
Tensor operator*(float scalar, const Tensor& t) {
    return t * scalar;
}